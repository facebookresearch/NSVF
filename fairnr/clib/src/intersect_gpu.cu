#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// 
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"


// input: ray_start (b, m, 3)
//        ray_dir (b, m, 3)
//        points (b, n, 3)
// output: idx (b, m, n_max)
//         min_d (b, m, n_max)
//         max_d (b, m, n_max)
__global__ void ball_intersect_point_kernel(
            int b, int n, int m, float radius,
            int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ points,
            int *__restrict__ idx,
            float *__restrict__ min_depth,
            float *__restrict__ max_depth) {

  int batch_index = blockIdx.x;
  points += batch_index * n * 3;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  min_depth += batch_index * m * n_max;
  max_depth += batch_index * m * n_max;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  float radius2 = radius * radius;

  for (int j = index; j < m; j += stride) {
    
    float x0 = ray_start[j * 3 + 0];
    float y0 = ray_start[j * 3 + 1];
    float z0 = ray_start[j * 3 + 2];
    float xw = ray_dir[j * 3 + 0];
    float yw = ray_dir[j * 3 + 1];
    float zw = ray_dir[j * 3 + 2];
    
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }

    for (int k = 0, cnt = 0; k < n && cnt < n_max; ++k) {
      float x = points[k * 3 + 0] - x0;
      float y = points[k * 3 + 1] - y0;
      float z = points[k * 3 + 2] - z0;
      float d2 = x * x + y * y + z * z;
      float d2_proj = pow(x * xw + y * yw + z * zw, 2);
      float r2 = d2 - d2_proj;
      
      if (r2 < radius2) {
        idx[j * n_max + cnt] = k;
        
        float depth = sqrt(d2_proj);
        float depth_delta = sqrt(radius2 - r2);
        
        min_depth[j * n_max + cnt] = depth - depth_delta;
        max_depth[j * n_max + cnt] = depth + depth_delta;
        ++cnt;
      }
    }
  }
}


__global__ void aabb_intersect_point_kernel(
            int b, int n, int m, float voxelsize,
            int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ points,
            int *__restrict__ idx,
            float *__restrict__ min_depth,
            float *__restrict__ max_depth) {
  
  int batch_index = blockIdx.x;
  points += batch_index * n * 3;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  min_depth += batch_index * m * n_max;
  max_depth += batch_index * m * n_max;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  float half_voxel = voxelsize * 0.5; 

  for (int j = index; j < m; j += stride) {
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }

    for (int k = 0, cnt = 0; k < n && cnt < n_max; ++k) {
      float f_low = 0;
      float f_high = 100000.;
      bool missed = false;
      
      for (int d = 0; d < 3; ++d) {
        
        float f_dim_low, f_dim_high, temp;
        float inv_ray_dir = 1.0 / ray_dir[j * 3 + d];
        float start = ray_start[j * 3 + d];
        float aabb = points[k * 3 + d];
      
        f_dim_low  = (aabb - half_voxel - start) * inv_ray_dir;
        f_dim_high = (aabb + half_voxel - start) * inv_ray_dir;
      
        // Make sure low is less than high
        if (f_dim_high < f_dim_low) {
          temp = f_dim_low;
          f_dim_low = f_dim_high;
          f_dim_high = temp;
        }

        // If this dimension's high is less than the low we got then we definitely missed.
        if (f_dim_high < f_low) {
          missed = true; 
          break;
        }
      
        // Likewise if the low is less than the high.
        if (f_dim_low > f_high) {
          missed = true; 
          break;
        }
          
        // Add the clip from this dimension to the previous results 
        f_low = (f_dim_low > f_low) ? f_dim_low : f_low;
        f_high = (f_dim_high < f_high) ? f_dim_high : f_high;
        
        if (f_low > f_high) {
          missed = true; 
          break;
        }
      }

      if (!missed){
        idx[j * n_max + cnt] = k;
        min_depth[j * n_max + cnt] = f_low;
        max_depth[j * n_max + cnt] = f_high;
        ++cnt;
      }
    }
  }
}

void ball_intersect_point_kernel_wrapper(
  int b, int n, int m, float radius, int n_max,
  const float *ray_start, const float *ray_dir, const float *points,
  int *idx, float *min_depth, float *max_depth) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ball_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, n_max, ray_start, ray_dir, points, idx, min_depth, max_depth);
  
  CUDA_CHECK_ERRORS();
}


void aabb_intersect_point_kernel_wrapper(
  int b, int n, int m, float voxelsize, int n_max,
  const float *ray_start, const float *ray_dir, const float *points,
  int *idx, float *min_depth, float *max_depth) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  aabb_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, voxelsize, n_max, ray_start, ray_dir, points, idx, min_depth, max_depth);
  
  CUDA_CHECK_ERRORS();
}


__global__ void uniform_ray_sampling_kernel(
            int b, int num_rays, 
            int max_hits,
            int max_steps,
            float step_size,
            const int *__restrict__ pts_idx,
            const float *__restrict__ min_depth,
            const float *__restrict__ max_depth,
            const float *__restrict__ uniform_noise,
            int *__restrict__ sampled_idx,
            float *__restrict__ sampled_depth,
            float *__restrict__ sampled_dists) {
  
  int batch_index = blockIdx.x;
  int index = threadIdx.x;
  int stride = blockDim.x;

  pts_idx += batch_index * num_rays * max_hits;
  min_depth += batch_index * num_rays * max_hits;
  max_depth += batch_index * num_rays * max_hits;

  uniform_noise += batch_index * num_rays * max_steps;
  sampled_idx += batch_index * num_rays * max_steps;
  sampled_depth += batch_index * num_rays * max_steps;
  sampled_dists += batch_index * num_rays * max_steps;

  // loop over all rays
  for (int j = index; j < num_rays; j += stride) {
    int H = j * max_hits, K = j * max_steps;
    int s = 0, ucur = 0, umin = 0, umax = 0;
    float last_min_depth, last_max_depth, curr_depth;
    
    // sort all depths
    while (true) {
      if (pts_idx[H + umax] == -1 || umax == max_hits || ucur == max_steps) {
        break;  // reach the maximum
      }
      if (umin < max_hits) {
        last_min_depth = min_depth[H + umin];
      }
      if (umax < max_hits) {
        last_max_depth = max_depth[H + umax];
      }
      if (ucur < max_steps) {
        curr_depth = min_depth[H] + (float(ucur) + uniform_noise[K + ucur]) * step_size;
      }
      
      if (last_max_depth <= curr_depth && last_max_depth <= last_min_depth) {
        sampled_depth[K + s] = last_max_depth;
        sampled_idx[K + s] = pts_idx[H + umax];
        umax++; s++; continue;
      }
      if (curr_depth <= last_min_depth && curr_depth <= last_max_depth) {
        sampled_depth[K + s] = curr_depth;
        sampled_idx[K + s] = pts_idx[H + umin - 1];
        ucur++; s++; continue;
      }
      if (last_min_depth <= curr_depth && last_min_depth <= last_max_depth) {
        sampled_depth[K + s] = last_min_depth;
        sampled_idx[K + s] = pts_idx[H + umin];
        umin++; s++; continue;
      }
    }

    float l_depth, r_depth;
    int step = 0;
    for (ucur = 0, umin = 0, umax = 0; ucur < max_steps - 1; ucur++) {
      l_depth = sampled_depth[K + ucur];
      r_depth = sampled_depth[K + ucur + 1];  
      sampled_depth[K + ucur] = (l_depth + r_depth) * .5;
      sampled_dists[K + ucur] = (r_depth - l_depth);
      if (sampled_depth[K + ucur] >= min_depth[H + umin] && umin < max_hits) umin++;
      if (sampled_depth[K + ucur] >= max_depth[H + umax] && umax < max_hits) umax++;
      if (umax == max_hits || pts_idx[H + umax] == -1) break;
      if (umin - 1 == umax && sampled_dists[K + ucur] > 0) {
        sampled_depth[K + step] = sampled_depth[K + ucur];
        sampled_dists[K + step] = sampled_dists[K + ucur];
        sampled_idx[K + step] = sampled_idx[K + ucur];
        step++;
      }
    }
    for (int s = step; s < max_steps; s++) {
      sampled_idx[K + s] = -1;
    }
  }


}


void uniform_ray_sampling_kernel_wrapper(
  int b, int num_rays, int max_hits, int max_steps, float step_size,
  const int *pts_idx, const float *min_depth, const float *max_depth, const float *uniform_noise,
  int *sampled_idx, float *sampled_depth, float *sampled_dists) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  uniform_ray_sampling_kernel<<<b, opt_n_threads(num_rays), 0, stream>>>(
      b, num_rays, max_hits, max_steps, step_size, pts_idx, 
      min_depth, max_depth, uniform_noise, sampled_idx, sampled_depth, sampled_dists);
  
  CUDA_CHECK_ERRORS();
}


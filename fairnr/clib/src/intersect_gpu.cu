#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// 
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "cutil_math.h"  // required for float3 vector math


__global__ void ball_intersect_point_kernel(
            int b, int n, int m, float radius,
            int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ points,
            int *__restrict__ idx,
            float *__restrict__ min_depth,
            float *__restrict__ max_depth) {

  int batch_index = blockIdx.x;
  points += batch_index * n * 3;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  min_depth += batch_index * m * n_max;
  max_depth += batch_index * m * n_max;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  float radius2 = radius * radius;

  for (int j = index; j < m; j += stride) {
    
    float x0 = ray_start[j * 3 + 0];
    float y0 = ray_start[j * 3 + 1];
    float z0 = ray_start[j * 3 + 2];
    float xw = ray_dir[j * 3 + 0];
    float yw = ray_dir[j * 3 + 1];
    float zw = ray_dir[j * 3 + 2];
    
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }

    for (int k = 0, cnt = 0; k < n && cnt < n_max; ++k) {
      float x = points[k * 3 + 0] - x0;
      float y = points[k * 3 + 1] - y0;
      float z = points[k * 3 + 2] - z0;
      float d2 = x * x + y * y + z * z;
      float d2_proj = pow(x * xw + y * yw + z * zw, 2);
      float r2 = d2 - d2_proj;
      
      if (r2 < radius2) {
        idx[j * n_max + cnt] = k;
        
        float depth = sqrt(d2_proj);
        float depth_blur = sqrt(radius2 - r2);
        
        min_depth[j * n_max + cnt] = depth - depth_blur;
        max_depth[j * n_max + cnt] = depth + depth_blur;
        ++cnt;
      }
    }
  }
}


__device__ float2 RayAABBIntersection(
  const float3 &ori,
  const float3 &dir,
  const float3 &center,
  float half_voxel) {

  float f_low = 0;
  float f_high = 100000.;
  float f_dim_low, f_dim_high, temp, inv_ray_dir, start, aabb;

  for (int d = 0; d < 3; ++d) {  
    switch (d) {
      case 0:
        inv_ray_dir = __fdividef(1.0f, dir.x); start = ori.x; aabb = center.x; break;
      case 1:
        inv_ray_dir = __fdividef(1.0f, dir.y); start = ori.y; aabb = center.y; break;
      case 2:
        inv_ray_dir = __fdividef(1.0f, dir.z); start = ori.z; aabb = center.z; break;
    }
  
    f_dim_low  = (aabb - half_voxel - start) * inv_ray_dir;
    f_dim_high = (aabb + half_voxel - start) * inv_ray_dir;
  
    // Make sure low is less than high
    if (f_dim_high < f_dim_low) {
      temp = f_dim_low;
      f_dim_low = f_dim_high;
      f_dim_high = temp;
    }

    // If this dimension's high is less than the low we got then we definitely missed.
    if (f_dim_high < f_low) {
      return make_float2(-1.0f, -1.0f);
    }
  
    // Likewise if the low is less than the high.
    if (f_dim_low > f_high) {
      return make_float2(-1.0f, -1.0f);
    }
      
    // Add the clip from this dimension to the previous results 
    f_low = (f_dim_low > f_low) ? f_dim_low : f_low;
    f_high = (f_dim_high < f_high) ? f_dim_high : f_high;
    
    if (f_low > f_high) {
      return make_float2(-1.0f, -1.0f);
    }
  }
  return make_float2(f_low, f_high);
}


__global__ void aabb_intersect_point_kernel(
            int b, int n, int m, float voxelsize,
            int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ points,
            int *__restrict__ idx,
            float *__restrict__ min_depth,
            float *__restrict__ max_depth) {
  
  int batch_index = blockIdx.x;
  points += batch_index * n * 3;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  min_depth += batch_index * m * n_max;
  max_depth += batch_index * m * n_max;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  float half_voxel = voxelsize * 0.5; 

  for (int j = index; j < m; j += stride) {
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }

    for (int k = 0, cnt = 0; k < n && cnt < n_max; ++k) {
      float2 depths = RayAABBIntersection(
        make_float3(ray_start[j * 3 + 0], ray_start[j * 3 + 1], ray_start[j * 3 + 2]),
        make_float3(ray_dir[j * 3 + 0], ray_dir[j * 3 + 1], ray_dir[j * 3 + 2]),
        make_float3(points[k * 3 + 0], points[k * 3 + 1], points[k * 3 + 2]),
        half_voxel);

      if (depths.x > -1.0f){
        idx[j * n_max + cnt] = k;
        min_depth[j * n_max + cnt] = depths.x;
        max_depth[j * n_max + cnt] = depths.y;
        ++cnt;
      }
    }
  }
}


__global__ void svo_intersect_point_kernel(
            int b, int n, int m, float voxelsize,
            int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ points,
            const int *__restrict__ children,
            int *__restrict__ idx,
            float *__restrict__ min_depth,
            float *__restrict__ max_depth) {
  /*
  TODO: this is an inefficient implementation of the 
        navie Ray -- Sparse Voxel Octree Intersection. 
        It can be further improved using:
        
        Revelles, Jorge, Carlos Urena, and Miguel Lastra. 
        "An efficient parametric algorithm for octree traversal." (2000).
  */
  int batch_index = blockIdx.x;
  points += batch_index * n * 3;
  children += batch_index * n * 9;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  min_depth += batch_index * m * n_max;
  max_depth += batch_index * m * n_max;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  float half_voxel = voxelsize * 0.5; 

  for (int j = index; j < m; j += stride) {
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }
    int stack[256] = {-1};    // DFS, initialize the stack
    int ptr = 0, cnt = 0, k = -1;
    stack[ptr] = n - 1;       // ROOT node is always the last
    while (ptr > -1 && cnt < n_max) {
      assert((ptr < 256));

      // evaluate the current node
      k = stack[ptr];
      float2 depths = RayAABBIntersection(
        make_float3(ray_start[j * 3 + 0], ray_start[j * 3 + 1], ray_start[j * 3 + 2]),
        make_float3(ray_dir[j * 3 + 0], ray_dir[j * 3 + 1], ray_dir[j * 3 + 2]),
        make_float3(points[k * 3 + 0], points[k * 3 + 1], points[k * 3 + 2]),
        half_voxel * float(children[k * 9 + 8]));
      stack[ptr] = -1; ptr--;

      if (depths.x > -1.0f) { // ray did not miss the voxel
        // TODO: here it should be able to know which children is ok, further optimize the code
        if (children[k * 9 + 8] == 1) {  // this is a terminal node 
          idx[j * n_max + cnt] = k;
          min_depth[j * n_max + cnt] = depths.x;
          max_depth[j * n_max + cnt] = depths.y;
          ++cnt; continue;
        }

        for (int u = 0; u < 8; u++) {
          if (children[k * 9 + u] > -1) {
            ptr++; stack[ptr] = children[k * 9 + u]; // push child to the stack
          }
        }  
      }
    }
  }
}


__device__ float3 RayTriangleIntersection(
  const float3 &ori,
  const float3 &dir,
	const float3 &v0,
	const float3 &v1,
  const float3 &v2,
  float blur) {
  
  float3 v0v1 = v1 - v0;
  float3 v0v2 = v2 - v0;
  float3 v0O = ori - v0;
  float3 dir_crs_v0v2 = cross(dir, v0v2);
  
  float det = dot(v0v1, dir_crs_v0v2);
  det = __fdividef(1.0f, det);  // CUDA intrinsic function 
  
	float u = dot(v0O, dir_crs_v0v2) * det;
	if (u < 0.0f - blur || u > 1.0f + blur)
		return make_float3(-1.0f, 0.0f, 0.0f);

  float3 v0O_crs_v0v1 = cross(v0O, v0v1);
	float v = dot(dir, v0O_crs_v0v1) * det;
	if (v < 0.0f - blur || v > 1.0f + blur)
    return make_float3(-1.0f, 0.0f, 0.0f);
    
  if ((u + v) < 0.0f - blur || (u + v) > 1.0f + blur)
    return make_float3(-1.0f, 0.0f, 0.0f);

  float t = dot(v0v2, v0O_crs_v0v1) * det;
  return make_float3(t, u, v);
}


__global__ void triangle_intersect_point_kernel(
            int b, int n, int m, float cagesize,
            float blur, int n_max,
            const float *__restrict__ ray_start,
            const float *__restrict__ ray_dir,
            const float *__restrict__ face_points,
            int *__restrict__ idx,
            float *__restrict__ depth,
            float *__restrict__ uv) {
  
  int batch_index = blockIdx.x;
  face_points += batch_index * n * 9;
  ray_start += batch_index * m * 3;
  ray_dir += batch_index * m * 3;
  idx += batch_index * m * n_max;
  depth += batch_index * m * n_max * 3;
  uv += batch_index * m * n_max * 2;
    
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int j = index; j < m; j += stride) {
    // go over rays
    for (int l = 0; l < n_max; ++l) {
      idx[j * n_max + l] = -1;
    }

    int cnt = 0;
    for (int k = 0; k < n && cnt < n_max; ++k) {
      // go over triangles
      float3 tuv = RayTriangleIntersection(
        make_float3(ray_start[j * 3 + 0], ray_start[j * 3 + 1], ray_start[j * 3 + 2]),
        make_float3(ray_dir[j * 3 + 0], ray_dir[j * 3 + 1], ray_dir[j * 3 + 2]),
        make_float3(face_points[k * 9 + 0], face_points[k * 9 + 1], face_points[k * 9 + 2]),
        make_float3(face_points[k * 9 + 3], face_points[k * 9 + 4], face_points[k * 9 + 5]),
        make_float3(face_points[k * 9 + 6], face_points[k * 9 + 7], face_points[k * 9 + 8]),
        blur);

      if (tuv.x > 0) {
        int ki = k;
        float d = tuv.x, u = tuv.y, v = tuv.z;

        // sort
        for (int l = 0; l < cnt; l++) {
          if (d < depth[j * n_max * 3 + l * 3]) {
            swap(ki, idx[j * n_max + l]);
            swap(d, depth[j * n_max * 3 + l * 3]);
            swap(u, uv[j * n_max * 2 + l * 2]);
            swap(v, uv[j * n_max * 2 + l * 2 + 1]);
          }
        }
        idx[j * n_max + cnt] = ki;
        depth[j * n_max * 3 + cnt * 3] = d;
        uv[j * n_max * 2 + cnt * 2] = u;
        uv[j * n_max * 2 + cnt * 2 + 1] = v;
        cnt++;
      }
    }

    for (int l = 0; l < cnt; l++) {
      // compute min_depth
      if (l == 0) 
        depth[j * n_max * 3 + l * 3 + 1] = -cagesize;
      else
        depth[j * n_max * 3 + l * 3 + 1] = -fminf(cagesize, 
          .5 * (depth[j * n_max * 3 + l * 3] - depth[j * n_max * 3 + l * 3 - 3]));
      
      // compute max_depth
      if (l == cnt - 1)
        depth[j * n_max * 3 + l * 3 + 2] = cagesize;
      else
        depth[j * n_max * 3 + l * 3 + 2] = fminf(cagesize, 
          .5 * (depth[j * n_max * 3 + l * 3 + 3] - depth[j * n_max * 3 + l * 3]));
    }
  }
}

void ball_intersect_point_kernel_wrapper(
  int b, int n, int m, float radius, int n_max,
  const float *ray_start, const float *ray_dir, const float *points,
  int *idx, float *min_depth, float *max_depth) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ball_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, n_max, ray_start, ray_dir, points, idx, min_depth, max_depth);
  
  CUDA_CHECK_ERRORS();
}


void aabb_intersect_point_kernel_wrapper(
  int b, int n, int m, float voxelsize, int n_max,
  const float *ray_start, const float *ray_dir, const float *points,
  int *idx, float *min_depth, float *max_depth) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  aabb_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, voxelsize, n_max, ray_start, ray_dir, points, idx, min_depth, max_depth);
  
  CUDA_CHECK_ERRORS();
}


void svo_intersect_point_kernel_wrapper(
  int b, int n, int m, float voxelsize, int n_max,
  const float *ray_start, const float *ray_dir, const float *points, const int *children,
  int *idx, float *min_depth, float *max_depth) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  svo_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, voxelsize, n_max, ray_start, ray_dir, points, children, idx, min_depth, max_depth);
  
  CUDA_CHECK_ERRORS();
}


void triangle_intersect_point_kernel_wrapper(
  int b, int n, int m, float cagesize, float blur, int n_max,
  const float *ray_start, const float *ray_dir, const float *face_points,
  int *idx, float *depth, float *uv) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  triangle_intersect_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, cagesize, blur, n_max, ray_start, ray_dir, face_points, idx, depth, uv);
  
  CUDA_CHECK_ERRORS();
}


__global__ void uniform_ray_sampling_kernel(
            int b, int num_rays, 
            int max_hits,
            int max_steps,
            float step_size,
            const int *__restrict__ pts_idx,
            const float *__restrict__ min_depth,
            const float *__restrict__ max_depth,
            const float *__restrict__ uniform_noise,
            int *__restrict__ sampled_idx,
            float *__restrict__ sampled_depth,
            float *__restrict__ sampled_dists) {
  
  int batch_index = blockIdx.x;
  int index = threadIdx.x;
  int stride = blockDim.x;

  pts_idx += batch_index * num_rays * max_hits;
  min_depth += batch_index * num_rays * max_hits;
  max_depth += batch_index * num_rays * max_hits;

  uniform_noise += batch_index * num_rays * max_steps;
  sampled_idx += batch_index * num_rays * max_steps;
  sampled_depth += batch_index * num_rays * max_steps;
  sampled_dists += batch_index * num_rays * max_steps;

  // loop over all rays
  for (int j = index; j < num_rays; j += stride) {
    int H = j * max_hits, K = j * max_steps;
    int s = 0, ucur = 0, umin = 0, umax = 0;
    float last_min_depth, last_max_depth, curr_depth;
    
    // sort all depths
    while (true) {
      if (umax == max_hits || ucur == max_steps || pts_idx[H + umax] == -1) {
        break;  // reach the maximum
      }
      if (umin < max_hits) {
        last_min_depth = min_depth[H + umin];
      }
      if (umax < max_hits) {
        last_max_depth = max_depth[H + umax];
      }
      if (ucur < max_steps) {
        curr_depth = min_depth[H] + (float(ucur) + uniform_noise[K + ucur]) * step_size;
      }
      
      if (last_max_depth <= curr_depth && last_max_depth <= last_min_depth) {
        sampled_depth[K + s] = last_max_depth;
        sampled_idx[K + s] = pts_idx[H + umax];
        umax++; s++; continue;
      }
      if (curr_depth <= last_min_depth && curr_depth <= last_max_depth) {
        sampled_depth[K + s] = curr_depth;
        sampled_idx[K + s] = pts_idx[H + umin - 1];
        ucur++; s++; continue;
      }
      if (last_min_depth <= curr_depth && last_min_depth <= last_max_depth) {
        sampled_depth[K + s] = last_min_depth;
        sampled_idx[K + s] = pts_idx[H + umin];
        umin++; s++; continue;
      }
    }

    float l_depth, r_depth;
    int step = 0;
    for (ucur = 0, umin = 0, umax = 0; ucur < max_steps - 1; ucur++) {
      if (sampled_idx[K + ucur + 1] == -1) break;
      l_depth = sampled_depth[K + ucur];
      r_depth = sampled_depth[K + ucur + 1];  
      sampled_depth[K + ucur] = (l_depth + r_depth) * .5;
      sampled_dists[K + ucur] = (r_depth - l_depth);
      if (umin < max_hits && sampled_depth[K + ucur] >= min_depth[H + umin] && pts_idx[H + umin] > -1) umin++;
      if (umax < max_hits && sampled_depth[K + ucur] >= max_depth[H + umax] && pts_idx[H + umax] > -1) umax++;
      if (umax == max_hits || pts_idx[H + umax] == -1) break;
      if (umin - 1 == umax && sampled_dists[K + ucur] > 0) {
        sampled_depth[K + step] = sampled_depth[K + ucur];
        sampled_dists[K + step] = sampled_dists[K + ucur];
        sampled_idx[K + step] = sampled_idx[K + ucur];
        step++;
      }
    }
    
    for (int s = step; s < max_steps; s++) {
      sampled_idx[K + s] = -1;
    }
  }
}


void uniform_ray_sampling_kernel_wrapper(
  int b, int num_rays, int max_hits, int max_steps, float step_size,
  const int *pts_idx, const float *min_depth, const float *max_depth, const float *uniform_noise,
  int *sampled_idx, float *sampled_depth, float *sampled_dists) {
  
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  uniform_ray_sampling_kernel<<<b, opt_n_threads(num_rays), 0, stream>>>(
      b, num_rays, max_hits, max_steps, step_size, pts_idx, 
      min_depth, max_depth, uniform_noise, sampled_idx, sampled_depth, sampled_dists);
  
  CUDA_CHECK_ERRORS();
}

